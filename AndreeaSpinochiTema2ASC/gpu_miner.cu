#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>
#define THREADS_PER_BLOCK 500

__global__ void findNonce(BYTE *block_content, BYTE *difficulty, uint64_t *found_nonce, BYTE *block_hash) {
	// calculate the nonce for the current block and current thread
    uint64_t nonce = blockIdx.x * blockDim.x + threadIdx.x;
    BYTE local_hash[SHA256_HASH_SIZE];
    char nonce_string[NONCE_SIZE];
    BYTE local_block_content[BLOCK_SIZE];
	// set the maximum nonce value
	uint64_t max_nonce = MAX_NONCE;

	// copy the block content to local block content
    memcpy(local_block_content, block_content, BLOCK_SIZE);
	// calculate the length of the local block content
    int content_length = d_strlen((const char*)local_block_content);

    // verifying if the nonce was already found by another thread
    if (*found_nonce > 0) {
        return;
    }

    // verify if the nonce is greater than the maximum nonce
    if (nonce > max_nonce) {
        return;
    }

    // generate the nonce string
    intToString(nonce, nonce_string);

    // concatenating the nonce string to the local block content
    d_strcpy((char*)local_block_content + content_length, nonce_string);

    // calculate the hash of the local block content
    apply_sha256(local_block_content, d_strlen((const char*)local_block_content), local_hash, 1);

    // compare the hash with the given difficulty
    if (compare_hashes(local_hash, difficulty) <= 0) {
        // if the hash is less than the difficulty, then the nonce is found
		// and then the local block hash and nonce are copied to the global block hash and nonce
		// using an atomic operation
		if(atomicExch((unsigned long long int*)found_nonce, nonce) == 0) {
			*found_nonce = nonce;
			memcpy(block_hash, local_hash, SHA256_HASH_SIZE);
		}
    }
}


int main(int argc, char **argv) {
    // Declarations
    BYTE block_content[BLOCK_SIZE];
    BYTE block_hash[SHA256_HASH_SIZE] = {0};
    uint64_t nonce = 0;
    size_t current_length;

    // Compute the top hash of transactions
    BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
         tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
         tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE];

    apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
    apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
    apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
    apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);

    strcpy((char *)tx12, (const char *)hashed_tx1);
    strcat((char *)tx12, (const char *)hashed_tx2);
    apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);

    strcpy((char *)tx34, (const char *)hashed_tx3);
    strcat((char *)tx34, (const char *)hashed_tx4);
    apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);

    strcpy((char *)tx1234, (const char *)hashed_tx12);
    strcat((char *)tx1234, (const char *)hashed_tx34);
    apply_sha256(tx1234, strlen((const char*)tx1234), top_hash, 1);

    // Prepare the initial content of the block by combining previous block hash and top hash
    strcpy((char *)block_content, (const char *)prev_block_hash);
    strcat((char *)block_content, (const char *)top_hash);
    current_length = strlen((char*) block_content);
    printf("Block content without nonce: %s\n", block_content);

    // Device memory for kernel
    BYTE *d_block_content, *d_difficulty, *d_found_block_hash;
    uint64_t *d_found_nonce;
    hipMalloc(&d_block_content, BLOCK_SIZE);
    hipMalloc(&d_difficulty, SHA256_HASH_SIZE);
    hipMalloc(&d_found_nonce, sizeof(uint64_t));
    hipMalloc(&d_found_block_hash, SHA256_HASH_SIZE);

    // Copy data to device
    hipMemcpy(d_block_content, block_content, current_length + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_difficulty, difficulty_5_zeros, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_found_block_hash, block_hash, current_length + 1, hipMemcpyHostToDevice);

    // Timing setup
    hipEvent_t start, stop;
    startTiming(&start, &stop);

    // Launch kernel
    findNonce<<<MAX_NONCE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_block_content, d_difficulty, d_found_nonce, d_found_block_hash);

    hipDeviceSynchronize();
    float seconds = stopTiming(&start, &stop);

    // Copy back the nonce and block hash
    hipMemcpy(&nonce, d_found_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(block_hash, d_found_block_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);

    // Print the result
    printResult(block_hash, nonce, seconds);

    // Free device memory
    hipFree(d_block_content);
    hipFree(d_difficulty);
    hipFree(d_found_nonce);
    hipFree(d_found_block_hash);

    return 0;
}
